#include "hip/hip_runtime.h"
#define BLOCKS_NUM 10
#define BLOCK_SIZE 256

#include <iostream>
using namespace std;

__global__ void my_kernel(float *in, float *out) {
int n = threadIdx.x + blockIdx.x * BLOCK_SIZE;
out[n] = in[n] * in[n]; // квадрат
// ...
}

int main() {
	float data[BLOCKS_NUM * BLOCK_SIZE]; //     CPU память
	for (int i=0; i<BLOCKS_NUM*BLOCK_SIZE; i+=BLOCK_SIZE)
	    data[i]=i;

	hipSetDevice(0); // выбор устройства
	float *in, *out; // GPU память

	// GPU выделение памяти
	uint memory_size = sizeof(float) * BLOCKS_NUM * BLOCK_SIZE;
	hipMalloc((void **)&in, memory_size);
	hipMalloc((void **)&out, memory_size);
	 //        копируем память на устройство
	hipMemcpy(in, data, memory_size, hipMemcpyHostToDevice);
	dim3 block(BLOCK_SIZE);
	dim3 grid(BLOCKS_NUM);

	//    запускаем ядро
	my_kernel<<<grid, block>>>(in, out);
	hipDeviceSynchronize(); //      ждём когда досчитается

	//          копируем результаты обратно на хост
	hipMemcpy(data, out, memory_size, hipMemcpyDeviceToHost);
	for (int i=0; i<BLOCKS_NUM*BLOCK_SIZE; i+=BLOCK_SIZE)
		std::cout<<data[i]<<" ";
	std::cout<<std::endl;

	// GPU! не забываем отчистить память на
	hipFree(in);
	hipFree(out);
	return 0;
}
